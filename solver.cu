#include "hip/hip_runtime.h"
#include "solver.cuh"

__global__ void find_best_move_kernel(int* best_move, const int* board) {
    // This kernel will evaluate all possible moves and store the best one in best_move
    // The implementation of this kernel is left as an exercise for the reader
}

int Solver::find_best_move(const TicTacToe& game) {
    int* d_best_move;
    hipMalloc(&d_best_move, sizeof(int));

    int* d_board;
    hipMalloc(&d_board, sizeof(int) * 9);
    hipMemcpy(d_board, game.board, sizeof(int) * 9, hipMemcpyHostToDevice);

    find_best_move_kernel<<<1, 9>>>(d_best_move, d_board);

    int best_move;
    hipMemcpy(&best_move, d_best_move, sizeof(int), hipMemcpyDeviceToHost);

    hipFree(d_best_move);
    hipFree(d_board);

    return best_move;
}
